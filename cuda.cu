
#include <hip/hip_runtime.h>
#include <cstdio>
#include <cmath>
#include <vector>
#include <chrono>
using namespace std;

#define TILE_SIZE 32

__global__ void matmul(const float* A, const float* B, float* C, int N)
{
    __shared__ float a[TILE_SIZE][TILE_SIZE + 1];
    __shared__ float b[TILE_SIZE][TILE_SIZE + 1];

    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int bx = blockIdx.x;
    int by = blockIdx.y;

    int Col = bx * TILE_SIZE + tx;
    int Row = by * TILE_SIZE + ty;

    float sum = 0.0f;
    
    for(int i = 0; i < N / TILE_SIZE; i++)
    {
        a[ty][tx] = A[Row * N + (i * TILE_SIZE + tx)];
        b[ty][tx] = B[Col + (i * TILE_SIZE + ty) * N];
        __syncthreads();

        for(int k = 0; k < TILE_SIZE; k++)
            sum += a[ty][k] * b[k][tx];
        __syncthreads();
    }

    C[Row * N + Col] = sum;
}

int main(int argc, char** argv) {
    const int N = 256;
    vector<float> A(N*N);
    vector<float> B(N*N);
    vector<float> C(N*N, 0);

    for (int i=0; i<N; i++) {
        for (int j=0; j<N; j++) {
            A[N*i+j] = drand48();
            B[N*i+j] = drand48();
        }
    }

    float *a, *b, *c;
    hipMalloc(&a, sizeof(float) * N * N);
    hipMalloc(&b, sizeof(float) * N * N);
    hipMalloc(&c, sizeof(float) * N * N);

    hipMemcpy(a,&A[0], sizeof(float) * N * N, hipMemcpyHostToDevice);
    hipMemcpy(b,&B[0], sizeof(float) * N * N, hipMemcpyHostToDevice);
    auto tic = chrono::steady_clock::now();

    
    dim3 grid(8,8);
    dim3 block(TILE_SIZE, TILE_SIZE);
    matmul<<<grid, block>>>(a, b, c, N);
    hipDeviceSynchronize();

    auto toc = chrono::steady_clock::now();
    double comp_time = chrono::duration<double>(toc - tic).count();

    hipMemcpy(&C[0], c, sizeof(float) * N * N, hipMemcpyDeviceToHost);

    for (int i=0; i<N; i++)
        for (int j=0; j<N; j++)
            for (int k=0; k<N; k++)
                C[N*i+j] -= A[N*i+k] * B[N*k+j];

    double err = 0;
    for (int i=0; i<N; i++)
        for (int j=0; j<N; j++)
            err += fabs(C[i*N+j]);

    printf("N    : %d\n",N);
    printf("comp : %lf s\n", comp_time);
    printf("total: %lf s (%lf GFlops)\n", comp_time, 2.*N*N*N/comp_time/1e9);
    printf("error: %lf\n",err/N/N);
}
